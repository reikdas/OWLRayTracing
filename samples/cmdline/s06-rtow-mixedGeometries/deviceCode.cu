#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2019-2020 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

// #include "deviceCode.h"
#include "GeomTypes.h"
#include <optix_device.h>
#include <bits/stdc++.h>
// here
// #include</home/min/a/nagara16/owl/owl/include/owl/common/parallel/parallel_for.h>

using namespace owl;

#define NUM_SAMPLES_PER_PIXEL 16

// here
__constant__ MyGlobals optixLaunchParams;
#define FLOAT_MIN 1.175494351e-38
#define FLOAT_MAX 3.402823466e+38
__device__ int intersections = 0;
// ==================================================================
// bounding box programs - since these don't actually use the material
// they're all the same irrespective of geometry type, so use a
// template ...
// ==================================================================
template <typename SphereGeomType>
inline __device__ void boundsProg(const void *geomData,
								  box3f &primBounds,
								  const int primID)
{
	const SphereGeomType &self = *(const SphereGeomType *)geomData;
	const Sphere sphere = self.prims[primID];
	primBounds = box3f()
					 .extend(sphere.center - self.rad)
					 .extend(sphere.center + self.rad);
}

OPTIX_BOUNDS_PROGRAM(Spheres)
(const void *geomData,
 box3f &primBounds,
 const int primID)
{
	boundsProg<SpheresGeom>(geomData, primBounds, primID);
}

// ==================================================================
// intersect programs - still all the same, since they don't use the
// material, either
// ==================================================================
OPTIX_INTERSECT_PROGRAM(Spheres) ()
{
	// ID of the sphere the ray intersected
	const int primID = optixGetPrimitiveIndex();

	// ID of the ray
	int xID = optixGetLaunchIndex().x;

	// The number of neighbors as sepcified by the user
	int k = optixLaunchParams.k;

	// Count number of intersections
	optixLaunchParams.frameBuffer[xID * k].intersections += 1;

	// Check if we have already processed this sphere in a previous iteration. If check == 1, we've seen it before
	int check = 0;
	for (int i = 0; i < k; i++) {
		if (optixLaunchParams.frameBuffer[xID * k + i].ind == primID) {
			// if(xID == 6)
			// printf("INTERSECT: Already intersected %d\n",primID);
			check = 1;
			break;
		}
	}

	if (check == 0) {

		// Access the array of spheres
		const SpheresGeom &selfs = owl::getProgramData<SpheresGeom>();

		// Access intersected sphere using its ID
		Sphere self = selfs.prims[primID];

		/*
		The frameBuffer is arranged as: [0,1,..,k-2,k-1, k,k+1..2k-1, 2k..3k-1], where neighbors of sphere 0 are stored at [0..k-1], sphere 1 at [k..2k-1] etc..
		The last element for each subgroup contains the neighbor at the maximum distance. 
		Example: for sphere 0, sphere k-1 is the most distant neighbor; for sphere 1, sphere 2k-1 etc..
		*/
		float maxDist = optixLaunchParams.frameBuffer[xID * k + k - 1].dist;

		// Avoid self-intersections where xID == primID as it will always have distance = 0
		if (xID != primID) {
			// Get coordinates of center of sphere (point in the original dataset)
			const vec3f org = optixGetWorldRayOrigin();
			float x, y, z;

			// Get x2-x1, y2-y1, z2-z1
			// Calculate Euclidean distance between ray and intersected object
			x = self.center.x - org.x;
			y = self.center.y - org.y;
			z = self.center.z - org.z;
			float distance = std::sqrt((x * x) + (y * y) + (z * z));

			// Check if distance to currently intersceted sphere is less than the max we have seen so far
			if (distance < maxDist) {
				// Smaller than max distance, so it will be a new neighbor
				if (optixLaunchParams.frameBuffer[xID * k].numNeighbors > 0)
					optixLaunchParams.frameBuffer[xID * k].numNeighbors -= 1;

				int q = 0, w = k - 1;
				for (; q < k; q++) {
					// Handle frameBuffer values from previous iterations: if the distance and index of neighbor is same => have already seen this before
					// Need to figure out where to insert the current point
					if (distance < optixLaunchParams.frameBuffer[xID * k + q].dist) 
						break;
				}
				// Update the neighbor list
				for (; w > q; w--) {
					optixLaunchParams.frameBuffer[xID * k + w].dist = optixLaunchParams.frameBuffer[xID * k + w - 1].dist;
					optixLaunchParams.frameBuffer[xID * k + w].ind = optixLaunchParams.frameBuffer[xID * k + w - 1].ind;
				}
				optixLaunchParams.frameBuffer[xID * k + w].dist = distance;
				optixLaunchParams.frameBuffer[xID * k + w].ind = primID;
			}
		}
	}
}

OPTIX_RAYGEN_PROGRAM(rayGen) ()
{

	const RayGenData &self = owl::getProgramData<RayGenData>();
	vec3f color = 0.f;
	int xID = optixGetLaunchIndex().x;
	int knn = optixLaunchParams.k;

	// Only launch rays if the point hasn't found k nearest neighbors yet. We use xID*k so that point 0 has neighbors from 0 to k-1 etc..
	if (optixLaunchParams.frameBuffer[xID * knn].numNeighbors > 0) {
		owl::Ray ray(optixLaunchParams.spheres[xID].center, vec3f(0, 0, 1), 0, 1.e-16f);
		owl::traceRay(self.world, ray, color);
	}
}
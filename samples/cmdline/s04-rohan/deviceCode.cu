#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2019-2020 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "GeomTypes.h"
#include <optix_device.h>
#include "bitmap.h"
#include <cmath>

//#include</home/min/a/nagara16/Downloads/owl/owl/include/owl/common/parallel/parallel_for.h>

using namespace owl;
__constant__ MyGlobals optixLaunchParams;

////////////////////////////////////////////////////////////////CODE BEGINS//////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ inline char getBitAtPositionInBitmap(char *bitmap, unsigned long position) {
  unsigned long bytePosition = position / 8;
  unsigned long bitPosition = position % 8;

  //int byte = bitmap[bytePosition];
  char bit = (bitmap[bytePosition] >> bitPosition) & 1;
  return bit;
}

__device__ inline u_int deviceSetBitAtPositionInBitmap(u_int *bitmap, unsigned long position, u_int value) {
  unsigned long bytePosition = position / 32;
  unsigned long bitPosition = position % 32;

  u_int *bytePtr = &bitmap[bytePosition];
  u_int bitToSet = (value << bitPosition);

  return atomicOr(bytePtr, bitToSet);
}

OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)()
{
  const TrianglesGeomData &self = owl::getProgramData<TrianglesGeomData>();
  const int primID = optixGetPrimitiveIndex();
  //printf("primID: %d\n", primID);
  PerRayData &prd = owl::getPRD<PerRayData>();

  // compute force between point and bhNode
  Point point = optixLaunchParams.devicePoints[prd.pointID];
  deviceBhNode bhNode = optixLaunchParams.deviceBhNodes[primID];

  optixLaunchParams.computedForces[prd.pointID] += (((point.mass * bhNode.mass)) / prd.r_2) * GRAVITATIONAL_CONSTANT;
  CustomRay rayObject;
  rayObject.primID = bhNode.autoRopePrimId;
  rayObject.orgin = bhNode.autoRopeRayLocation;
  rayObject.pointID = prd.pointID;
  prd.rayToLaunch = rayObject;

  // if(prd.pointID == 0) {
  // printf("%sIntersected yay!%s\n",
  //          OWL_TERMINAL_GREEN,
  //          OWL_TERMINAL_DEFAULT);
  // printf("Current rtComputedForce: %f\n", optixLaunchParams.computedForces[prd.pointID]);
  // }
}

OPTIX_MISS_PROGRAM(miss)()
{
  const MissProgData &self = owl::getProgramData<MissProgData>();
  // printf("%sMissed it!%s\n",
  //          OWL_TERMINAL_RED,
  //          OWL_TERMINAL_DEFAULT);
  PerRayData &prd = owl::getPRD<PerRayData>();

  deviceBhNode bhNode = optixLaunchParams.deviceBhNodes[prd.primID];;
  
  if(bhNode.isLeaf == 1) {
    optixLaunchParams.computedForces[prd.pointID] += (((optixLaunchParams.devicePoints[prd.pointID].mass * bhNode.mass)) / prd.r_2) * GRAVITATIONAL_CONSTANT;
    // if(prd.pointID == 0) {
    // printf("%sHit leaf in miss yay!%s\n",
    //        OWL_TERMINAL_GREEN,
    //        OWL_TERMINAL_DEFAULT); }
  } else {
    //printf("PrimID: %d\n", prd.primID);
  }
  CustomRay rayObject;
  rayObject.primID = bhNode.nextPrimId;
  rayObject.pointID = prd.pointID;
  rayObject.orgin = bhNode.nextRayLocation;
  prd.rayToLaunch = rayObject;
  prd.rayEnd = 0;
  //atomicAdd(optixLaunchParams.raysToLaunch, bhNode.numChildren);

  //printf("Ray distance %f.\n", prd.r_2);

}

OPTIX_RAYGEN_PROGRAM(rayGen)()
{
  const RayGenData &self = owl::getProgramData<RayGenData>();
  const vec2i pixelID = owl::getLaunchIndex();

  CustomRay currentRay = self.primaryLaunchRays[pixelID.x];
  Point point = optixLaunchParams.devicePoints[currentRay.pointID];
  deviceBhNode bhNode = optixLaunchParams.deviceBhNodes[currentRay.primID];

  // Calculate distance between point and bh node
  float dx = fabs(point.x - bhNode.centerOfMassX);
  float dy = fabs(point.y - bhNode.centerOfMassY);

  PerRayData prd;
  prd.r_2 = (dx * dx) + (dy * dy);
  prd.pointID = currentRay.pointID;
  prd.primID = currentRay.primID;
  prd.rayEnd = 0;
  //prd.insertIndex = 0;
  float rayLength = sqrtf(prd.r_2) * 0.5f;
  // if(prd.pointID == 0) printf("Num prims %d\n", optixLaunchParams.numPrims);
  // if(prd.pointID == 0) printf("Index: %d | PrimID: %d | rayLength: %f\n", 0, prd.primID, rayLength);

  // Launch rays
  int index = 0;
  owl::Ray ray(currentRay.orgin, vec3f(1,0,0), 0, rayLength);
  while(prd.rayEnd == 0) {
    if(rayLength != 0.0f) {
      owl::traceRay(self.world, ray, prd);
    } else {
      CustomRay rayObject;
      rayObject.primID = bhNode.nextPrimId;
      rayObject.pointID = prd.pointID;
      rayObject.orgin = bhNode.nextRayLocation;
      prd.rayToLaunch = rayObject;
    }

    currentRay = prd.rayToLaunch;
    bhNode = optixLaunchParams.deviceBhNodes[currentRay.primID];

    dx = point.x - bhNode.centerOfMassX;
    dy = point.y - bhNode.centerOfMassY;
    prd.r_2 = (dx * dx) + (dy * dy);
    prd.primID = currentRay.primID;
    rayLength = sqrtf(prd.r_2) * 0.5f;

    ray.origin = currentRay.orgin;
    ray.tmax = rayLength;
    if(prd.primID >= optixLaunchParams.numPrims || prd.primID == 0) {
      prd.rayEnd = 1;
    }
    index++;
    // if(prd.pointID == 0) {
    //   printf("Index: %d | PrimID: %d | rayLength: %f | Origin: (%f, %f)\n", index, prd.primID, rayLength, ray.origin.x, ray.origin.y);
    //   //printf("insertIndex: %d\n", prd.insertIndex);
    // }
  }
}


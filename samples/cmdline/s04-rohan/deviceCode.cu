#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2019-2020 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "GeomTypes.h"
#include <optix_device.h>
#include "bitmap.h"
#include <cmath>

//#include</home/min/a/nagara16/Downloads/owl/owl/include/owl/common/parallel/parallel_for.h>

using namespace owl;
__constant__ MyGlobals optixLaunchParams;

////////////////////////////////////////////////////////////////CODE BEGINS//////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)()
{
  const TrianglesGeomData &self = owl::getProgramData<TrianglesGeomData>();
  const int primID = optixGetPrimitiveIndex();
  //printf("primID: %d\n", primID);
  PerRayData &prd = owl::getPRD<PerRayData>();

  // compute force between point and bhNode
  Point point = optixLaunchParams.devicePoints[prd.pointID];
  deviceBhNode bhNode = optixLaunchParams.deviceBhNodes[primID];

  //if(prd.pointID == 0) printf("Current rtComputedForce: %f\n", optixLaunchParams.computedForces[prd.pointID]);
  optixLaunchParams.computedForces[prd.pointID] += (((point.mass * bhNode.mass)) / prd.r_2) * GRAVITATIONAL_CONSTANT;
  // if(prd.pointID == 8124) {
  //     prd.result.didIntersect = 1;
  //     prd.result.isLeaf = 0;
  //     optixLaunchParams.intersectionResults[prd.result.index] = prd.result;
  //   }
  CustomRay rayObject;
  rayObject.primID = bhNode.autoRopePrimId;
  rayObject.orgin = bhNode.autoRopeRayLocation;
  rayObject.pointID = prd.pointID;
  prd.rayToLaunch = rayObject;
  //if(prd.pointID == 5382) printf("Approximated at node with mass! ->%f\n", bhNode.mass);
  // if(prd.pointID == 8124) {
  // printf("%sIntersected yay!%s\n",
  //          OWL_TERMINAL_GREEN,
  //          OWL_TERMINAL_DEFAULT);
  // // printf("Current rtComputedForce: %f\n", optixLaunchParams.computedForces[prd.pointID]);
  // }
}

OPTIX_MISS_PROGRAM(miss)()
{
  const MissProgData &self = owl::getProgramData<MissProgData>();
  // printf("%sMissed it!%s\n",
  //          OWL_TERMINAL_RED,
  //          OWL_TERMINAL_DEFAULT);
  PerRayData &prd = owl::getPRD<PerRayData>();

  deviceBhNode bhNode = optixLaunchParams.deviceBhNodes[prd.primID];;
  
  if(bhNode.isLeaf == 1) {
    optixLaunchParams.computedForces[prd.pointID] += (((optixLaunchParams.devicePoints[prd.pointID].mass * bhNode.mass)) / prd.r_2) * GRAVITATIONAL_CONSTANT;
    // if(prd.pointID == 8124) {
    //   prd.result.didIntersect = 0;
    //   prd.result.isLeaf = 1;
    //   optixLaunchParams.intersectionResults[prd.result.index] = prd.result;
    // }
  //if(prd.pointID == 5382) printf("Intersected leaf at node with mass! ->%f\n", bhNode.mass);
    // if(prd.pointID == 8124) {
    // printf("%sHit leaf in miss yay!%s\n",
    //        OWL_TERMINAL_GREEN,
    //        OWL_TERMINAL_DEFAULT); }
  } else {
    // if(prd.pointID == 8124) {
    //   prd.result.didIntersect = 0;
    //   prd.result.isLeaf = 0;
    //   optixLaunchParams.intersectionResults[prd.result.index] = prd.result;
    //   //printf("insertIndex: %d\n", prd.result.index);
    // }
    //printf("PrimID: %d\n", prd.primID);
  }
  CustomRay rayObject;
  rayObject.primID = bhNode.nextPrimId;
  rayObject.pointID = prd.pointID;
  rayObject.orgin = bhNode.nextRayLocation;
  prd.rayToLaunch = rayObject;
  prd.rayEnd = 0;
  //atomicAdd(optixLaunchParams.raysToLaunch, bhNode.numChildren);

  //printf("Ray distance %f.\n", prd.r_2);

}

OPTIX_RAYGEN_PROGRAM(rayGen)()
{
  const RayGenData &self = owl::getProgramData<RayGenData>();
  const vec2i pixelID = owl::getLaunchIndex();

  CustomRay currentRay = self.primaryLaunchRays[pixelID.x];
  Point point = optixLaunchParams.devicePoints[currentRay.pointID];
  deviceBhNode bhNode = optixLaunchParams.deviceBhNodes[currentRay.primID];

  // Calculate distance between point and bh node
  float dx = fabs(point.x - bhNode.centerOfMassX);
  float dy = fabs(point.y - bhNode.centerOfMassY);

  PerRayData prd;
  prd.r_2 = (dx * dx) + (dy * dy);
  prd.pointID = currentRay.pointID;
  prd.primID = currentRay.primID;
  prd.rayEnd = 0;
  //prd.insertIndex = 0;
  float rayLength = sqrtf(prd.r_2) * 0.5f;
  //if(prd.pointID == 0) printf("Num prims %d\n", optixLaunchParams.numPrims);
  //if(prd.pointID == 5382) printf("Index: %d | PrimID: %d | Mass: %f | rayLength: %f\n", 0, prd.primID, bhNode.mass, rayLength);

  // Launch rays
  int index = 0;
  prd.index = index;
  prd.rayLength = rayLength;
  owl::Ray ray(currentRay.orgin, vec3f(1,0,0), 0, rayLength);
  while(prd.rayEnd == 0) {
    if(rayLength != 0.0f) {
      owl::traceRay(self.world, ray, prd);
    } else {
      CustomRay rayObject;
      rayObject.primID = bhNode.nextPrimId;
      rayObject.pointID = prd.pointID;
      rayObject.orgin = bhNode.nextRayLocation;
      prd.rayToLaunch = rayObject;
    }

    currentRay = prd.rayToLaunch;
    bhNode = optixLaunchParams.deviceBhNodes[currentRay.primID];

    dx = point.x - bhNode.centerOfMassX;
    dy = point.y - bhNode.centerOfMassY;
    prd.r_2 = (dx * dx) + (dy * dy);
    prd.primID = currentRay.primID;
    rayLength = sqrtf(prd.r_2) * 0.5;

    ray.origin = currentRay.orgin;
    ray.tmax = rayLength;
    if(prd.primID >= optixLaunchParams.numPrims || prd.primID == 0) {
      prd.rayEnd = 1;
    }
    index++;
    prd.index = index;
    prd.rayLength = rayLength;
    // if(prd.pointID == 8124) {
    //   IntersectionResult result;
    //   result.index = index;
    //   result.primID = prd.primID;
    //   result.mass = bhNode.mass;
    //   result.rayLength = rayLength;
    //   //result.didIntersect = 1;
    //   prd.result = result;
    //   //optixLaunchParams.intersectionResults[prd.index] = result;
    // }
    // if(prd.pointID == 5382) {
    //   //printf("Index: %d | PrimID: %d | Mass: %f | rayLength: %f | Origin: (%f, %f)\n", index, prd.primID, bhNode.mass, rayLength, ray.origin.x, ray.origin.y);
    // }
  }
}


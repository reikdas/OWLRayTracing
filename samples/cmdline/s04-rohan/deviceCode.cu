#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2019-2020 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "GeomTypes.h"
#include <optix_device.h>
//#include</home/min/a/nagara16/Downloads/owl/owl/include/owl/common/parallel/parallel_for.h>

using namespace owl;
__constant__ MyGlobals optixLaunchParams;

////////////////////////////////////////////////////////////////CODE BEGINS//////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


// ==================================================================
// bounding box programs - since these don't actually use the material
// they're all the same irrespective of geometry type, so use a
// template ...
// ==================================================================
template<typename SphereGeomType>
inline __device__ void boundsProg(const void *geomData,
                                  box3f &primBounds,
                                  const int primID)
{   
  const SphereGeomType &self = *(const SphereGeomType*)geomData;
	//printf("Radius = %f\n",self.rad);
  const Sphere sphere = self.prims[primID];
  primBounds = box3f()
    .extend(sphere.center - self.rad)
    .extend(sphere.center + self.rad);
}

OPTIX_BOUNDS_PROGRAM(Spheres)(const void  *geomData,
                                        box3f       &primBounds,
                                        const int    primID)
{ boundsProg<SpheresGeom>(geomData,primBounds,primID); }

// ==================================================================
// intersect programs - still all the same, since they don't use the
// material, either
// ==================================================================
OPTIX_INTERSECT_PROGRAM(Spheres)()
{ 
	const int primID = optixGetPrimitiveIndex();
	int xID = optixGetLaunchIndex().x;
  int level = optixGetLaunchIndex().y;
  if(optixLaunchParams.parallelLaunch == 0) {
    level = optixLaunchParams.yIDx;
  }
  const SpheresGeom &selfs = owl::getProgramData<SpheresGeom>();
  Sphere self = selfs.prims[primID];
  float &prd = getPRD<float>();
  
  
  //Inside circle?
  //const vec3f org = optixGetWorldRayOrigin();
  // float x,y,z;
  
  //Get closest hit triangle's associated circle
  // x = self.center.x - org.x;
  // y = self.center.y - org.y;
  // z = self.center.z - org.z;

  if(self.isLeaf == false) 
    printf("Ray %d in level %d intersected circle with center x = %f, y = %f, z = %f , mass = %f, color = %f\n", xID, level, self.center.x, self.center.y, self.center.z, self.mass, prd);
}

// ==================================================================
// miss and raygen
// ==================================================================

OPTIX_RAYGEN_PROGRAM(rayGen)()
{
	const RayGenData &self = owl::getProgramData<RayGenData>();
	float level = 1.0f;  
	int xID = optixGetLaunchIndex().x;
  int yID = optixGetLaunchIndex().y;
	owl::Ray ray(vec3f(self.points[xID].x,self.points[xID].y,0), vec3f(0,0,1), 0, 1.e-16f);
  //printf("Level: %d \n", optixLaunchParams.yIDx);
  //printf("Starting ray for level %d with index: %d\n", yID, xID);
  //printf("Starting ray in level %d at circle with center x = %f, y = %f \n", yID, self.points[xID].x, self.points[xID].y);
  if(optixLaunchParams.parallelLaunch == 0) {
    yID = optixLaunchParams.yIDx;
  }
  //printf("Starting ray for level %d with index: %d\n", yID, xID);
  owl::traceRay(self.worlds[yID], ray, level);
}


#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2019-2020 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "GeomTypes.h"
#include <optix_device.h>
#include "bitmap.h"
//#include</home/min/a/nagara16/Downloads/owl/owl/include/owl/common/parallel/parallel_for.h>

using namespace owl;
__constant__ MyGlobals optixLaunchParams;

////////////////////////////////////////////////////////////////CODE BEGINS//////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


// ==================================================================
// bounding box programs - since these don't actually use the material
// they're all the same irrespective of geometry type, so use a
// template ...
// ==================================================================
template<typename SphereGeomType>
inline __device__ void boundsProg(const void *geomData,
                                  box3f &primBounds,
                                  const int primID)
{   
  const SphereGeomType &self = *(const SphereGeomType*)geomData;
	//printf("Radius = %f\n",self.rad);
  const Sphere sphere = self.prims[primID];
  primBounds = box3f()
    .extend(sphere.center - self.rad)
    .extend(sphere.center + self.rad);
}

OPTIX_BOUNDS_PROGRAM(Spheres)(const void  *geomData,
                                        box3f       &primBounds,
                                        const int    primID)
{ boundsProg<SpheresGeom>(geomData,primBounds,primID); }

// ==================================================================
// intersect programs - still all the same, since they don't use the
// material, either
// ==================================================================

// __device__ inline char atomicOr(char *bitmap, unsigned long position, char value) {
//   unsigned long bytePosition = position / 8;
//   unsigned long bitPosition = position % 8;

//   char *address = &bitmap[bytePosition];
//   char const long_address_modulo = reinterpret_cast< size_t >( address ) & 0x3;
//   u_int *const base_address = reinterpret_cast< u_int * >( reinterpret_cast< size_t >( address ) - long_address_modulo ); 
//   //printf("Address of x: is %p with offset: %d with base address: %p\n", address, long_address_modulo, base_address);
//   u_int constexpr byteSelection[] = {0x3214, 0x3240, 0x3410, 0x4210};
//   u_int const byteSelector = byteSelection[long_address_modulo];
//   u_int long_old = *base_address;
//   u_int long_assumed, long_val, replacement;

//   do {
//     long_assumed = long_old;

//     long_val = static_cast<u_int>((value << bitPosition) | long_old);
//     replacement = __byte_perm(long_old, long_val, byteSelector);
//     printf("Position: (%lu,%lu), long_old: %x and long_val: %x and replacement: %x\n", bytePosition, bitPosition, long_old, long_val, replacement);
//     //printf("base_address: %x and long_assumed: %x\n", *base_address, long_assumed);
//     long_old = ::atomicCAS(base_address, long_assumed, replacement);
//     printf("Position: (%lu,%lu), base_address: %x and long_old: %x and long_assumed: %x\n", bytePosition, bitPosition, *base_address, long_old, long_assumed);

//   } while (long_old != long_assumed);

//   return long_old;
// }

__device__ inline char getBitAtPositionInBitmap(char *bitmap, unsigned long position) {
  unsigned long bytePosition = position / 8;
  unsigned long bitPosition = position % 8;

  //int byte = bitmap[bytePosition];
  char bit = (bitmap[bytePosition] >> bitPosition) & 1;
  return bit;
}

__device__ inline u_int deviceSetBitAtPositionInBitmap(u_int *bitmap, unsigned long position, u_int value) {
  unsigned long bytePosition = position / 32;
  unsigned long bitPosition = position % 32;

  u_int *bytePtr = &bitmap[bytePosition];
  u_int bitToSet = (value << bitPosition);

  return atomicOr(bytePtr, bitToSet);
}

OPTIX_INTERSECT_PROGRAM(Spheres)()
{ 
	const int primID = optixGetPrimitiveIndex();
	int xID = optixGetLaunchIndex().x;
  int level = optixGetLaunchIndex().y + 1;
  if(optixLaunchParams.parallelLaunch == 0) {
    level = optixLaunchParams.yIDx;
  }
  const SpheresGeom &selfs = owl::getProgramData<SpheresGeom>();
  Sphere self = selfs.prims[primID];
  float radius = selfs.rad;
  PerRayData &prd = owl::getPRD<PerRayData>();
  
  
  //Inside circle?
  const vec3f org = optixGetWorldRayOrigin();
  float x,y,z;
  
  //Get closest hit triangle's associated circle
  x = self.center.x - org.x;
  y = self.center.y - org.y;
  z = self.center.z - org.z;

  long *nodesPerLevel = optixLaunchParams.nodesPerLevel;
  //int *offsetPerLevel = optixLaunchParams.offsetPerLevel;
  //printf("index: %d\n", ((xID * nodesPerLevel[level]) + primID));
  if(std::sqrt((x*x) + (y*y) + (z*z)) <= radius)
	{
    //printf("Ray %d in level %d with %lu nodes intersected primID: %d \n", xID, level, nodesPerLevel[level] , primID);
    u_int result = deviceSetBitAtPositionInBitmap(optixLaunchParams.outputIntersectionData, ((xID * nodesPerLevel[level]) + primID), 1);
    //char[0] = 0x11;
    //char[1] = 0x01;
    //char[2] = 0x00;
    //char[3] = 0x10;
    // int[0] = 0x11010010;
    // unsigned int test1 = static_cast<unsigned int>(test);
    // unsigned int test2 = 0x9ABCDEF0;
    // unsigned int selector = 0x3210;
    //uint32_t result = __byte_perm(test2, test1, selector);
    //printf("Before: %02x\n", static_cast<unsigned char>(optixLaunchParams.outputIntersectionData[0]));
    //char result = atomicOr(optixLaunchParams.outputIntersectionData, ((xID * nodesPerLevel[level]) + primID), 1);
    //printf("After: %02x\n", static_cast<unsigned char>(optixLaunchParams.outputIntersectionData[0]));
    //printf("Byte position: %lu and value %d,  Bit position: %lu and value %d\n", bytePosition, byte, bitPosition, bit);
    // unsigned long bytePosition = ((xID * nodesPerLevel[level]) + primID) / 8;
    // unsigned long bitPosition = ((xID * nodesPerLevel[level]) + primID) % 8;

    // char byte = optixLaunchParams.outputIntersectionData[bytePosition];
    // char bit = (byte >> bitPosition) & 1;
    // if(bit != 1) {
    //  printf("Byte position: %lu and value %d,  Bit position: %lu and value %d\n", bytePosition, byte, bitPosition, bit);
    //   //optixLaunchParams.outputIntersectionData[bytePosition] == (1 << bitPosition);
    //   optixLaunchParams.outputIntersectionData[bytePosition] = 1;
    // }
    //optixLaunchParams.outputIntersectionData[((xID * nodesPerLevel[level]) + primID)] = 1;
    //printf("Ray %d in level %d intersected circle with center x = %f, y = %f, z = %f , mass = %f\n", xID, level, self.center.x, self.center.y, self.center.z, self.mass);
  }
  //printf("At idx: %d, Value of bitmap: %d\n", ((xID * nodesPerLevel[level]) + primID), getBitAtPositionInBitmap(optixLaunchParams.outputIntersectionData, ((xID * nodesPerLevel[level]) + primID)));
    
}

// ==================================================================
// miss and raygen
// ==================================================================

OPTIX_RAYGEN_PROGRAM(rayGen)()
{
	const RayGenData &self = owl::getProgramData<RayGenData>();
	int xID = optixGetLaunchIndex().x + optixLaunchParams.xIDxOffset; 
  int yID = optixGetLaunchIndex().y + 1;
	owl::Ray ray(vec3f(self.points[xID].x,self.points[xID].y,0), vec3f(0,0,1), 0, 1.e-16f);
  PerRayData prd;

  if(optixLaunchParams.parallelLaunch == 0) {
    yID = optixLaunchParams.yIDx;
  }

  owl::traceRay(self.worlds[yID], ray, prd);
  
}

